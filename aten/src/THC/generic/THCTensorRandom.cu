
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorRandom.cu"
#else

#include "ATen/cuda/CUDAContext.h"

#define NUM_BLOCKS min((int)THCCeilDiv(size, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)


void THCTensor_(logNormal)(THCState* state, THCTensor *self_, double mean, double stdv)
{

  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generateLogNormal<scalar_t><<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, mean, stdv);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(exponential)(THCState* state, THCTensor *self_, double lambda)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_exponential<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, lambda);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(cauchy)(THCState* state, THCTensor *self_, double median, double sigma)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_cauchy<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, median, sigma);

  THCTensor_(freeCopyTo)(state, self, self_);
};

void THCTensor_(renormRows)(struct THCState* state,
                             THCTensor* t) {
  THAssert(THCTensor_(nDimensionLegacyAll)(state, t) == 2);
  int64_t rows = THCTensor_(size)(state, t, 0);
  int64_t cols = THCTensor_(size)(state, t, 1);

  cudaDeviceProp* props = at::cuda::getCurrentDeviceProperties();
  THAssert(props != NULL);

  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;

  dim3 grid(rows < numSM * 4 ? rows : numSM * 4);
  dim3 block(cols < maxThreads ? cols : maxThreads);

  renormRowsL1<scalar_t>
    <<<grid, block, block.x * sizeof(scalar_t),
    THCState_getCurrentStream(state)>>>(THCTensor_(data)(state, t),
                                        rows, cols);
}

void THCTensor_(multinomial)(struct THCState *state,
                              THCudaLongTensor *self,
                              THCTensor *prob_dist,
                              int n_sample,
                              int with_replacement)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, prob_dist));
  THCGenerator* gen = THCRandom_getGenerator(state);

  int inputSize = THCTensor_(nDimensionLegacyAll)(state, prob_dist);
  THArgCheck(inputSize > 0 && inputSize <= 2, 2,
             "prob_dist must be 1 or 2 dim");

  // Categories are in the innermost dimension
  int64_t numDist =
    inputSize == 1 ? 1 : THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 0);
  int64_t numCategoriesLong =
    inputSize == 1 ? THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 0) :
    THCTensor_(sizeLegacyNoScalars)(state, prob_dist, 1);

  // Since the index tensor is float, numCategories cannot exceed max
  // float integer precision
  THArgCheck(numCategoriesLong <= FLOAT32_MAX_CONSECUTIVE_INT, 2,
             "number of categories cannot exceed 2^24");
  int numCategories = (int) numCategoriesLong;

  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");

  if (!with_replacement) {
    THArgCheck(n_sample <= numCategories, 2,
               "cannot sample n_sample > prob_dist:size(1) samples without "
               "replacement");
  }

  int free_prob_dist = 0;

  // Restructure data for 2d
  if (inputSize == 1) {
    THCTensor *temp = THCTensor_(new)(state);
    THCTensor_(unsqueeze1d)(state, temp, prob_dist, 0);
    prob_dist = temp;
    free_prob_dist = 1;
  }

  THCudaLongTensor_resize2d(state, self, numDist, n_sample);

  // get current device properties
  cudaDeviceProp* props = at::cuda::getCurrentDeviceProperties();
  THAssert(props != NULL);
  int numSM = props->multiProcessorCount;
  int maxThreads = props->maxThreadsPerBlock;
  int maxShared = props->sharedMemPerBlock;
  int requiredShared = (numCategories < maxThreads ? numCategories : maxThreads)
                                * (sizeof(scalar_t) + sizeof(accreal));

  if (n_sample == 1 && maxShared >= requiredShared) {
    // Optimized allocation-free implementation
    // To exploit greater parallelism for the sampling, generate the
    // Uniform random samples in a separate kernel launch, into
    // temporarily allocated memory. The device RNG is thread-limited
    THCTensor *sampled = THCTensor_(newWithSize2d)(state, numDist, n_sample);
    auto out = THTensor_wrap(sampled);
    at::native::uniform_cuda_(out, 0.0, 1.0);

    dim3 block(numCategories < maxThreads ? numCategories : maxThreads);
    dim3 grid(numDist < numSM * 4 ? numDist : numSM * 4);

    sampleMultinomialOnce<scalar_t, accreal>
      <<<grid, block,
         requiredShared,
         THCState_getCurrentStream(state)>>>(
      THCudaLongTensor_data(state, self),
      numDist,
      numCategories,
      THCTensor_(data)(state, sampled),
      THCTensor_(data)(state, prob_dist),
      THCTensor_(stride)(state, prob_dist, 0),
      THCTensor_(stride)(state, prob_dist, 1)
      );
    THCTensor_(free)(state, sampled);
  } else {
    // Generic, slow implementation with memory allocations

    // For sampling without replacement, we modify the distribution
    // for subsequent samples in this space
    THCTensor* origDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, origDist, prob_dist);
    THCTensor_(copy)(state, origDist, prob_dist);

    THCTensor* normDist = THCTensor_(new)(state);
    THCTensor_(resizeAs)(state, normDist, prob_dist);

    THCTensor* prefixSum = THCTensor_(new)(state);

    // Renorm along rows
    THCTensor_(copy)(state, normDist, origDist);
    THCTensor_(renormRows)(state, normDist);

    // Prefix sum along rows
    THCTensor_(cumsum)(state, prefixSum, normDist, 1);

    if (with_replacement) {
      // Sample with replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from one
      // distribution concurrently.
      dim3 grid(numDist < MAX_NUM_BLOCKS ? numDist : MAX_NUM_BLOCKS);

      sampleMultinomialWithReplacement
        <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
          gen->state.gen_states,
          n_sample,
          THCudaLongTensor_data(state, self),
          numDist, numCategories,
          THCTensor_(data)(state, prefixSum),
          THCTensor_(data)(state, normDist));
    } else {
      // Sample without replacement

      // Binary search is warp divergent (so effectively we're running
      // with just a single thread), but for better utilization,
      // we need each block to have at least 4 warps.
      dim3 block(32, 4);

      // Each warp in a block will generate a sample from a different
      // distribution concurrently.
      ptrdiff_t numBlocks = THCCeilDiv(numDist, (int64_t) 4);
      dim3 grid(numBlocks < MAX_NUM_BLOCKS ? numBlocks : MAX_NUM_BLOCKS);

      for (int sample = 0; sample < n_sample; ++sample) {
        if (sample > 0) {
          // Update probabilities
          // Renorm along rows
          THCTensor_(copy)(state, normDist, origDist);
          THCTensor_(renormRows)(state, normDist);

          // Prefix sum along rows
          THCTensor_(cumsum)(state, prefixSum, normDist, 1);
        }

        // The kernel can only draw one sample before we have to
        // recalculate our distribution
        sampleMultinomialWithoutReplacement
          <<<grid, block, 0, THCState_getCurrentStream(state)>>>(
            gen->state.gen_states,
            n_sample,
            sample,
            THCudaLongTensor_data(state, self),
            numDist, numCategories,
            THCTensor_(data)(state, origDist),
            THCTensor_(data)(state, prefixSum));
      }
    }

    THCTensor_(free)(state, prefixSum);
    THCTensor_(free)(state, normDist);
    THCTensor_(free)(state, origDist);
  }

  // Revert data restructuring based on input sizes
  if (inputSize == 1) {
    THCudaLongTensor_resize1d(state, self, n_sample);
  }
  if (free_prob_dist) {
    THCTensor_(free)(state, prob_dist);
  }
}

void THCTensor_(multinomialAliasSetup)(THCState *state, THCTensor *_probs, THCudaLongTensor *_J, THCTensor *_q){
  THArgCheck(_probs->dim() == 1, 1,
             "expected 1-D probability tensor, got %d-D probability tensor instead",
             _probs->dim());
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THCTensor *probs = THCTensor_(newContiguous)(state, _probs);
  THAssert(THCTensor_(isContiguous)(state, probs));
  int64_t inputsize = THCTensor_(nElement)(state, probs);
  THCudaLongTensor *smaller = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *smaller_short = THCudaLongTensor_newWithSize1d(state, inputsize);
  THCudaLongTensor *larger_short = THCudaLongTensor_newWithSize1d(state, inputsize);

  THCudaLongTensor_resize1d(state, _J, inputsize);
  THCTensor_(resize1d)(state, _q, inputsize);

  scalar_t one = ScalarConvert<int64_t, scalar_t>::to(1);
  int inputBlockDim = THCCeilDiv((int)inputsize + BLOCK_SIZE - 1, BLOCK_SIZE);
  aliasMultinomialFilter
    <<<inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state) >>>(
                     THCTensor_(data)(state, _q),
                     THCTensor_(data)(state, probs),
                     THCudaLongTensor_data(state, smaller),
                     THCudaLongTensor_data(state, larger),
                     THCudaLongTensor_data(state, _J),
                     THCudaLongTensor_data(state, smaller_short),
                     THCudaLongTensor_data(state, larger_short),
                     one, inputsize
                     );

  THCudaLongTensor_nonzero(state, smaller_short, smaller);
  THCudaLongTensor_nonzero(state, larger_short, larger);
  int h_large_c = THCudaLongTensor_nElement(state, larger_short);
  THCudaLongTensor_resize1d(state, smaller_short, inputsize);
  THCudaLongTensor_resize1d(state, larger_short, inputsize);
  aliasMultinomialSetup
    <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
                THCudaLongTensor_data(state, _J),
                THCTensor_(data)(state, _q),
                inputsize,
                THCudaLongTensor_data(state, smaller_short),
                THCudaLongTensor_data(state, larger_short),
                inputsize - h_large_c, h_large_c
                );
  scalar_t q_max = THCTensor_(maxall)(state, _q);
  condDiv<<<
    inputBlockDim, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
                      THCTensor_(data)(state, _q),
                      THCudaLongTensor_data(state, _J),
                      inputsize, q_max
                      );

  THCudaLongTensor_free(state, smaller);
  THCudaLongTensor_free(state, larger);
  THCudaLongTensor_free(state, smaller_short);
  THCudaLongTensor_free(state, larger_short);
  THCTensor_free(state, probs);
}

void THCTensor_(multinomialAliasDraw)(THCState *state, THCudaLongTensor *self, THCTensor *_q, THCudaLongTensor *_J, int n_sample){
  THArgCheck(_q->dim() == 1, 1,
             "expected 1-D probability table, got %d-D probability table instead",
             _q->dim());
  THArgCheck(_J->dim() == 1, 2,
             "expected 1-D alias table, got %d-D alias table instead",
             _J->dim());
  THArgCheck(n_sample > 0, 3, "cannot sample <= 0 samples");
  THAssert(THCTensor_(isContiguous)(state, _q));
  THAssert(THCudaLongTensor_isContiguous(state, _J));
  THCGenerator* gen = THCRandom_getGenerator(state);
  int64_t K = THCudaLongTensor_nElement(state, _J);
  THCudaLongTensor_resize1d(state, self, n_sample);
  ptrdiff_t size = THCudaLongTensor_nElement(state, self);

  THCTensor *uniform = THCTensor_(newWithSize1d)(state, n_sample);
  THCTensor *bernoulli = THCTensor_(newWithSize1d)(state, n_sample);

  auto out_uniform = THTensor_wrap(uniform);
  auto out_bernoulli = THTensor_wrap(bernoulli);
  at::native::uniform_cuda_(out_uniform, 0, K);
  at::native::uniform_cuda_(out_bernoulli, 0, 1);

  multinomialAliasDrawKernel
    <<<THCCeilDiv((int)n_sample+BLOCK_SIZE-1, BLOCK_SIZE), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
          size,
          THCudaLongTensor_data(state, self),
          THCudaLongTensor_data(state, _J),
          THCTensor_(data)(state, _q),
          K,
          THCTensor_(data)(state, uniform),
          THCTensor_(data)(state, bernoulli)
          );
  THCTensor_(free)(state, uniform);
  THCTensor_(free)(state, bernoulli);
}

#endif

#if defined(THC_REAL_IS_DOUBLE)
GENERATE_KERNEL1(generate_geometric, double, double p, double, curand_uniform_double, ceil(log(x) / log(1-p)))
#else
GENERATE_KERNEL1(generate_geometric, scalar_t, double p, float, curand_uniform, (ScalarConvert<float, scalar_t>::to(ceilf(logf(x) / log(1-p)))))
#endif

void THCTensor_(geometric)(THCState* state, THCTensor *self_, double p)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 1, self_));
  ptrdiff_t size = THCTensor_(nElement)(state, self_);
  if (size == 0) return;
  THCGenerator* gen = THCRandom_getGenerator(state);

  THCTensor *self = THCTensor_(newContiguous)(state, self_);
  scalar_t *data = THCTensor_(data)(state, self);

  generate_geometric<<<NUM_BLOCKS, BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
      gen->state.gen_states, size, data, p);

  THCTensor_(freeCopyTo)(state, self, self_);
};

#undef NUM_BLOCKS

#endif
